﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>

__device__ void user_program(FileSystem *fs, uchar *input, uchar *output) {
	
	/*
	/////////////// Test Case 1  ///////////////
	u32 fp = fs_open(fs, "t.txt\0", G_WRITE);
	fs_write(fs, input, 64, fp);
	fp = fs_open(fs, "b.txt\0", G_WRITE);
	fs_write(fs, input + 32, 32, fp);
	fp = fs_open(fs, "t.txt\0", G_WRITE);
	fs_write(fs, input + 32, 32, fp);
	fp = fs_open(fs, "t.txt\0", G_READ);
	fs_read(fs, output, 32, fp);
	fs_gsys(fs,LS_D);
	fs_gsys(fs, LS_S);
	fp = fs_open(fs, "b.txt\0", G_WRITE);
	fs_write(fs, input + 64, 12, fp);
	fs_gsys(fs, LS_S);
	fs_gsys(fs, LS_D);
	fs_gsys(fs, RM, "t.txt\0");
	fs_gsys(fs, LS_S);
	*/
	//printf("test\n");
	/*
	fs_gsys(fs, MKDIR, "soft\0");
	fs_gsys(fs, CD, "soft\0");
	fs_gsys(fs, MKDIR, "you\0");
	fs_gsys(fs, MKDIR, "app\0");
	fs_gsys(fs, CD, "app\0");
	fs_gsys(fs,PWD);
	fs_gsys(fs,CD_P);
	fs_gsys(fs,PWD);
	fs_gsys(fs,CD_P);
	fs_gsys(fs,PWD);
	printf("done\n");
	*/
	

	/*
	printf("%d\n",fs->volume[3]);
	u32 fp = fs_open(fs, "t.txt\0", G_WRITE);
	printf("%d\n",fp-0x00ff0000);
	printf("%d\n",fs->volume[4096+(fp-0x00ff0000)*32+31]);
	fp = fs_open(fs, "b.txt\0", G_WRITE);
	printf("%d\n",fp-0x00ff0000);
	fp = fs_open(fs, "b.txt\0", G_WRITE);
	printf("%d\n",fp-0x00ff0000);
	fp = fs_open(fs, "b.txt\0", G_READ);
	printf("%d\n",fp-0xff000000);
	int a=18;
	printf("%d\n",a/5);
	*/

	/*
	/////////////// Test Case 2  ///////////////
	u32 fp = fs_open(fs, "t.txt\0", G_WRITE);
	fs_write(fs,input, 64, fp);
	fp = fs_open(fs,"b.txt\0", G_WRITE);
	fs_write(fs,input + 32, 32, fp);
	fp = fs_open(fs,"t.txt\0", G_WRITE);
	fs_write(fs,input + 32, 32, fp);
	fp = fs_open(fs,"t.txt\0", G_READ);
	fs_read(fs,output, 32, fp);
	fs_gsys(fs,LS_D);
	fs_gsys(fs,LS_S);
	fp = fs_open(fs,"b.txt\0", G_WRITE);
	fs_write(fs,input + 64, 12, fp);
	fs_gsys(fs,LS_S);
	fs_gsys(fs,LS_D);
	fs_gsys(fs,RM, "t.txt\0");
	fs_gsys(fs,LS_S);
	char fname[10][20];
	for (int i = 0; i < 10; i++)
	{
		fname[i][0] = i + 33;
		for (int j = 1; j < 19; j++)
			fname[i][j] = 64 + j;
		fname[i][19] = '\0';
	}

	for (int i = 0; i < 10; i++)
	{
		fp = fs_open(fs,fname[i], G_WRITE);
		fs_write(fs,input + i, 24 + i, fp);
	}

	fs_gsys(fs,LS_S);

	for (int i = 0; i < 5; i++)
		fs_gsys(fs,RM, fname[i]);

	fs_gsys(fs,LS_D);
	*/
	/*
	/////////////// Test Case 3  ///////////////
	u32 fp = fs_open(fs, "t.txt\0", G_WRITE);
	fs_write(fs, input, 64, fp);
	fp = fs_open(fs, "b.txt\0", G_WRITE);
	fs_write(fs, input + 32, 32, fp);
	fp = fs_open(fs, "t.txt\0", G_WRITE);
	fs_write(fs, input + 32, 32, fp);
	fp = fs_open(fs, "t.txt\0", G_READ);
	fs_read(fs, output, 32, fp);
	fs_gsys(fs, LS_D);
	fs_gsys(fs, LS_S);
	fp = fs_open(fs, "b.txt\0", G_WRITE);
	fs_write(fs, input + 64, 12, fp);
	fs_gsys(fs, LS_S);
	fs_gsys(fs, LS_D);
	fs_gsys(fs, RM, "t.txt\0");
	fs_gsys(fs, LS_S);

	char fname[10][20];
	for (int i = 0; i < 10; i++)
	{
		fname[i][0] = i + 33;
		for (int j = 1; j < 19; j++)
			fname[i][j] = 64 + j;
		fname[i][19] = '\0';
	}

	for (int i = 0; i < 10; i++)
	{
		fp = fs_open(fs, fname[i], G_WRITE);
		fs_write(fs, input + i, 24 + i, fp);
	}

	fs_gsys(fs, LS_S);

	for (int i = 0; i < 5; i++)
		fs_gsys(fs, RM, fname[i]);

	fs_gsys(fs, LS_D);

	char fname2[1018][20];
	int p = 0;

	for (int k = 2; k < 15; k++)
		for (int i = 50; i <= 126; i++, p++)
		{
			fname2[p][0] = i;
			for (int j = 1; j < k; j++)
				fname2[p][j] = 64 + j;
			fname2[p][k] = '\0';
		}

	for (int i = 0; i < 1001; i++)
	{
		fp = fs_open(fs, fname2[i], G_WRITE);
		fs_write(fs, input + i, 24 + i, fp);
	}

	fs_gsys(fs, LS_S);
	fp = fs_open(fs, fname2[1000], G_READ);
	fs_read(fs, output + 1000, 1024, fp);

	char fname3[17][3];
	for (int i = 0; i < 17; i++)
	{
		fname3[i][0] = 97 + i;
		fname3[i][1] = 97 + i;
		fname3[i][2] = '\0';
		fp = fs_open(fs, fname3[i], G_WRITE);
		fs_write(fs, input + 1024 * i, 1024, fp);
	}

	fp = fs_open(fs, "EA\0", G_WRITE);
	fs_write(fs, input + 1024 * 100, 1024, fp);
	fs_gsys(fs, LS_S);
	*/

	/////////////// Test Case 4  ///////////////
	/*
    u32 fp = fs_open(fs, "32-block-0", G_WRITE);
    fs_write(fs, input, 99, fp);
    for (int j = 0; j < 1023; ++j) {
        char tag[] = "1024-block-????";
        int i = j;
        tag[11] = static_cast<char>(i / 1000 + '0');
        i = i % 1000;
        tag[12] = static_cast<char>(i / 100 + '0');
        i = i % 100;
        tag[13] = static_cast<char>(i / 10 + '0');
        i = i % 10;
        tag[14] = static_cast<char>(i + '0');
        fp = fs_open(fs, tag, G_WRITE);
        fs_write(fs, input + j * 1024, 1024, fp);
    }
	
    fs_gsys(fs, RM, "32-block-0");
    // now it has one 32byte at first, 1023 * 1024 file in the middle

    fp = fs_open(fs, "1024-block-1023", G_WRITE);
    printf("triggering gc\n");
    fs_write(fs, input + 1023 * 1024, 1024, fp);


    fs_gsys(fs, LS_D);
    for (int j = 0; j < 1024; ++j) {
        char tag[] = "1024-block-????";
        int i = j;
        tag[11] = static_cast<char>(i / 1000 + '0');
        i = i % 1000;
        tag[12] = static_cast<char>(i / 100 + '0');
        i = i % 100;
        tag[13] = static_cast<char>(i / 10 + '0');
        i = i % 10;
        tag[14] = static_cast<char>(i + '0');
        fp = fs_open(fs, tag, G_READ);
        fs_read(fs, output + j * 1024, 1024, fp);
    }
	*/
	
	//bonus
	u32 fp = fs_open(fs, "t.txt\0", G_WRITE);
	fs_write(fs, input, 64, fp);
	fp = fs_open(fs, "b.txt\0", G_WRITE);
	fs_write(fs, input + 32, 32, fp);
	fp = fs_open(fs, "t.txt\0", G_WRITE);
	fs_write(fs, input + 32, 32, fp);
	fp = fs_open(fs, "t.txt\0", G_READ);
	fs_read(fs, output, 32, fp);
	fs_gsys(fs, LS_D);
	fs_gsys(fs, LS_S);
	fs_gsys(fs, MKDIR, "app\0");
	fs_gsys(fs, LS_D);
	fs_gsys(fs, LS_S);
	fs_gsys(fs, CD, "app\0");
	fs_gsys(fs, LS_S);
	fp = fs_open(fs, "a.txt\0", G_WRITE);
	fs_write(fs, input + 128, 64, fp);
	fp = fs_open(fs, "b.txt\0", G_WRITE);
	fs_write(fs, input + 256, 32, fp);
	fs_gsys(fs, MKDIR, "soft\0");
	fs_gsys(fs, LS_S);
	fs_gsys(fs, LS_D);
	fs_gsys(fs, CD, "soft\0");
	fs_gsys(fs, PWD);
	fp = fs_open(fs, "A.txt\0", G_WRITE);
	fs_write(fs, input + 256, 64, fp);
	fp = fs_open(fs, "B.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "C.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "D.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fs_gsys(fs, LS_S);

	//fs_gsys(fs, RM, "A.txt\0");
	//fs_gsys(fs, LS_S);

	fs_gsys(fs, CD_P);
	fs_gsys(fs, LS_S);
	fs_gsys(fs, PWD);
	fs_gsys(fs, CD_P);
	fs_gsys(fs, LS_S);
	fs_gsys(fs, CD, "app\0");
	fs_gsys(fs, RM_RF, "soft\0");
	fs_gsys(fs, LS_S);
	fs_gsys(fs, CD_P);
	fs_gsys(fs, LS_S);
	
	/*
	//bonus tes2
	u32 fp = fs_open(fs, "a.txt\0", G_WRITE);
	fs_write(fs, input, 64, fp);
	fp = fs_open(fs, "b.txt\0", G_WRITE);
	fs_write(fs, input + 32, 32, fp);
	fp = fs_open(fs, "c.txt\0", G_WRITE);
	fs_write(fs, input + 32, 32, fp);
	fs_gsys(fs, MKDIR, "app\0");
	fs_gsys(fs, LS_S);
	//fs_gsys(fs, CD, "app\0");
	//fs_gsys(fs, CD_P);
	//fs_gsys(fs, LS_S);
	
	fs_gsys(fs, CD, "app\0");
	fp = fs_open(fs, "c.txt\0", G_WRITE);
	fs_write(fs, input + 32, 64, fp);
	fs_gsys(fs, MKDIR, "soft\0");
	fs_gsys(fs, LS_S);
	fs_gsys(fs, CD, "soft\0");
	fp = fs_open(fs, "d.txt\0", G_WRITE);
	fs_write(fs, input + 32, 32, fp);
	fp = fs_open(fs, "e.txt\0", G_WRITE);
	fs_write(fs, input + 32, 32, fp);
	fs_gsys(fs, LS_S);
	fs_gsys(fs, CD_P);
	fs_gsys(fs, LS_S);
	fs_gsys(fs, CD_P);
	fp = fs_open(fs, "f.txt\0", G_WRITE);
	fs_write(fs, input + 32, 32, fp);
	fs_gsys(fs, PWD);
	fs_gsys(fs, LS_D);
	fs_gsys(fs, LS_S);
	fs_gsys(fs, CD, "app\0");
	fs_gsys(fs, RM_RF, "soft\0");
	fs_gsys(fs, CD_P);
	fs_gsys(fs, LS_D);
	fs_gsys(fs, LS_S);
	fs_gsys(fs, CD, "app\0");
	fs_gsys(fs, MKDIR, "soft\0");
	fs_gsys(fs, LS_S);
	fs_gsys(fs, CD, "soft\0");
	fp = fs_open(fs, "a.txt\0", G_WRITE);
	fs_write(fs, input, 64, fp);
	fp = fs_open(fs, "b.txt\0", G_WRITE);
	fs_write(fs, input + 32, 32, fp);
	fp = fs_open(fs, "c.txt\0", G_WRITE);
	fs_write(fs, input + 32, 32, fp);
	fs_gsys(fs, LS_S);
	fs_gsys(fs, PWD);
	fs_gsys(fs, CD_P);
	fs_gsys(fs, LS_S);
	fs_gsys(fs, PWD);
	fs_gsys(fs, CD_P);
	fs_gsys(fs, PWD);
	fs_gsys(fs, LS_S);
	//fs_gsys(fs, RM_RF, "app\0");
	fs_gsys(fs, CD, "app\0");
	fs_gsys(fs, LS_S);
	fs_gsys(fs, RM, "c.txt\0");
	fs_gsys(fs, LS_S);
	fs_gsys(fs, CD, "soft\0");
	fs_gsys(fs, LS_S);
	fs_gsys(fs, RM, "a.txt\0");
	fs_gsys(fs, LS_S);
	//fs_gsys(fs, CD, "app\0");
	*/
	/*
	//bonus test3
	u32 fp = fs_open(fs, "1.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "2.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "3.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "4.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "5.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "6.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "7.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "8.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "9.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "10.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "11.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "12.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "13.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "14.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "15.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "16.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "17.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "18.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "19.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "20.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "21.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "22.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "23.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "24.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "25.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "26.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "27.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "28.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "29.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "30.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "31.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "32.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "33.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "34.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "35.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "36.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "37.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "38.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "39.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "40.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "41.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "42.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "43.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "44.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "45.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "46.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "47.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "48.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "49.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fs_gsys(fs, MKDIR, "soft\0");
	fs_gsys(fs, LS_S);
	fs_gsys(fs, CD, "soft\0");
	fp = fs_open(fs, "1.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "2.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "3.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "4.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "5.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "6.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "7.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "8.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "9.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "10.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "11.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "12.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "13.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "14.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "15.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "16.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "17.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "18.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "19.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "20.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "21.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "22.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "23.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "24.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "25.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "26.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "27.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "28.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "29.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "30.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "31.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "32.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "33.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "34.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "35.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "36.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "37.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "38.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "39.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "40.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "41.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "42.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "43.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "44.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "45.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "46.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "47.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "48.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "49.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fp = fs_open(fs, "50.txt\0", G_WRITE);
	fs_write(fs, input + 256, 1024, fp);
	fs_gsys(fs, LS_S);
	fs_gsys(fs, CD_P);
	fs_gsys(fs, LS_S);
	fs_gsys(fs, LS_D);
	fs_gsys(fs, RM_RF, "soft\0");
	fs_gsys(fs, LS_S);
	*/
}
